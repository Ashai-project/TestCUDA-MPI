/**
 * @file LoopSend4.cu
 * @author Ashai-project
 * @brief GPUDirect RDMA
 * cudaデバイス上のメモリ領域からcudaデバイスのメモリ領域へRDMA
 * @date 2024-05-10
 *
 */
#include <cstdio>
#include <iostream>
#include <mpi.h>
#include <hip/hip_runtime.h>
#if !defined(OPEN_MPI) || !OPEN_MPI
#error This source code uses an Open MPI-specific extension
#endif

/* Needed for MPIX_Query_cuda_support(), below */
#include <mpi-ext.h>
#define N 10

int main(int argc, char **argv)
{
    char hostname[256];
    int mpisize, mpirank, gpusize, gpurank, len;
    MPI_Init(&argc, &argv);
    // check cuda aware
    printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support())
    {
        printf("This MPI library has CUDA-aware support.\n");
    }
    else
    {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else  /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    MPI_Comm_size(MPI_COMM_WORLD, &mpisize);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpirank);
    MPI_Get_processor_name(hostname, &len);
    hipGetDeviceCount(&gpusize);
    hipSetDevice(mpirank % gpusize);
    hipGetDevice(&gpurank);
    int *recieve_buff_d,*recieve_buff_h;
    int recv_from;
    printf("Hostname    : %s\n", hostname);
    printf("MPI rank    : %d / %d  GPU device : %d / %d\n",
                   mpirank, mpisize, gpurank, gpusize);
    hipMalloc((void **)&recieve_buff_d, sizeof(int) * N);
    hipHostMalloc((void **)&recieve_buff_h, sizeof(int) * N);
    hipDeviceSynchronize();
    recv_from = mpirank - 4;
    for (int iroop = 0; iroop < 1000; iroop++)
    {
            MPI_Request request[1];
            MPI_Irecv(recieve_buff_d, N, MPI_INT, recv_from, 0, MPI_COMM_WORLD, &request[0]);
            MPI_Waitall(1, request, MPI_STATUS_IGNORE);
    }
    MPI_Finalize();
    hipMemcpy(recieve_buff_h, recieve_buff_d, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("MPI rank    : %d / %d RValue : %d\n", mpirank, mpisize, recieve_buff_h[0]);
}
