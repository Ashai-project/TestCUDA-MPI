    
    
    int ngpus;
    hipGetDeviceCount(&ngpus);
    
    for (int igpu = 0; igpu < ngpus; igpu++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, igpu);
        printf("Using Device %d : %s\n", igpu, deviceProp.name);

        // igpuのデバイスでデバイスコードkernelが実行される
        hipSetDevice(igpu);
        kernel<<<grid, block>>>(...);
    }
