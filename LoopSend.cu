#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <mpi.h>
#if !defined(OPEN_MPI) || !OPEN_MPI
#error This source code uses an Open MPI-specific extension
#endif

/* Needed for MPIX_Query_cuda_support(), below */
#include <mpi-ext.h>

__global__ void GPU_Kernel()
{
    printf(" GPU block  : %d / %d  GPU thread : %d / %d\n",
           blockIdx.x, gridDim.x, threadIdx.x, blockDim.x);
}

int main(int argc, char **argv)
{
    char hostname[256];
    int mpisize, mpirank, gpusize, gpurank, len;
    MPI_Init(&argc, &argv);
    //check cuda aware 
    printf("Compile time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    } else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    MPI_Comm_size(MPI_COMM_WORLD, &mpisize);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpirank);
    MPI_Get_processor_name(hostname, &len);
    hipGetDeviceCount(&gpusize);
    hipSetDevice(mpirank % gpusize);
    hipGetDevice(&gpurank);
    for (int irank = 0; irank < mpisize; irank++)
    {
        MPI_Barrier(MPI_COMM_WORLD);
        if (mpirank == irank)
        {
            printf("Hostname    : %s\n", hostname);
            printf("MPI rank    : %d / %d  GPU device : %d / %d\n",
                   mpirank, mpisize, gpurank, gpusize);
            // GPU_Kernel<<<2, 2>>>();
            int *send_b_d, *recieve_b_d;
            hipMalloc((void **)&send_b_d, sizeof(int) * 10);
            hipMalloc((void **)&recieve_b_d, sizeof(int) * 10);
            hipMemset(send_b_d, 0, sizeof(int) * 10);
            printf("success memset!\n");
            hipDeviceSynchronize();
            int recv_from = (mpirank + 1) % mpisize;
            int send_to = (mpirank - 1 + mpisize) % mpisize;
            printf("MPI rank : %d send: %d recieve: %d\n", mpirank, send_to, recv_from);
            MPI_Request request[2];
            MPI_Isend(send_b_d, 10, MPI_INT, send_to, 0, MPI_COMM_WORLD, &request[0]);
            MPI_Irecv(recieve_b_d, 10, MPI_INT, recv_from, 0, MPI_COMM_WORLD, &request[1]);
            MPI_Waitall(2, request, MPI_STATUS_IGNORE);
            int *recieve_b_h;
            hipHostMalloc((void **)&recieve_b_h, sizeof(int) * 10);
            hipMemcpy((void *)recieve_b_h, (void *)recieve_b_d, sizeof(int) * 10, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            printf("GPU device : %d / %d RValue : %d\n", gpurank, gpusize, recieve_b_h[0]);
        }
    }
    MPI_Finalize();
}
